#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------*/
/* Program: p2pBandwidthTest                                             */
/* Original code: https://github.com/NVIDIA/cuda-samples                 */
/*                                                                       */
/* This program measures GPU memory transfer rates in GB/s for simple.   */
/*-----------------------------------------------------------------------*/

#include <cstdio>
#include <vector>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

using namespace std;

typedef enum {
  P2P_WRITE = 0,
  P2P_READ = 1,
} P2PDataTransfer;

typedef enum {
  CE = 0,
  SM = 1,
} P2PEngine;

P2PEngine p2p_mechanism = CE;  // By default use Copy Engine

extern "C" double* testBandwidthMatrix(int numElems, int numGPUs, bool p2p, 
    P2PDataTransfer p2p_method, unsigned int nWarmups, unsigned int nRepeats);
extern "C" double* testBidirectionalBandwidthMatrix(int numElems, int numGPUs, bool p2p,
    unsigned int nWarmups, unsigned int nRepeats);
extern "C" double* testLatencyMatrix(int numGPUs, bool p2p, P2PDataTransfer p2p_method,
    unsigned int nWarmups, unsigned int nRepeats);

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError()                                       \
  {                                                            \
    hipError_t e = hipGetLastError();                        \
    if (e != hipSuccess) {                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(e));                           \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }

__global__ void delay(volatile int *flag,
                      unsigned long long timeout_clocks = 10000000) {
  // Wait until the application notifies us that it has completed queuing up the
  // experiment, or timeout and exit, allowing the application to make progress
  long long int start_clock, sample_clock;
  start_clock = clock64();

  while (!*flag) {
    sample_clock = clock64();

    if (sample_clock - start_clock > timeout_clocks) {
      break;
    }
  }
}

// This kernel is for demonstration purposes only, not a performant kernel for
// p2p transfers.
__global__ void copyp2p(int4 *__restrict__ dest, int4 const *__restrict__ src,
                        size_t num_elems) {
  size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  size_t gridSize = blockDim.x * gridDim.x;

#pragma unroll(5)
  for (size_t i = globalId; i < num_elems; i += gridSize) {
    dest[i] = src[i];
  }
}

void performP2PCopy(int *dest, int destDevice, int *src, int srcDevice,
                    int num_elems, int repeat, bool p2paccess,
                    hipStream_t streamToRun) {
  int blockSize = 0;
  int numBlocks = 0;

  hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, copyp2p);
  cudaCheckError();

  if (p2p_mechanism == SM && p2paccess) {
    for (int r = 0; r < repeat; r++) {
      copyp2p<<<numBlocks, blockSize, 0, streamToRun>>>(
          (int4 *)dest, (int4 *)src, num_elems / 4);
    }
  } else {
    for (int r = 0; r < repeat; r++) {
      hipMemcpyPeerAsync(dest, destDevice, src, srcDevice,
                          sizeof(int) * num_elems, streamToRun);
    }
  }
}

extern "C" double* testBandwidthMatrix(int numElems, int numGPUs, bool p2p, P2PDataTransfer p2p_method,
                                       unsigned int nWarmups, unsigned int nRepeats) {
  volatile int *flag = NULL;
  vector<int *> buffers(numGPUs);
  vector<int *> buffersD2D(numGPUs);  // buffer for D2D, that is, intra-GPU copy
  vector<hipEvent_t> start(numGPUs);
  vector<hipEvent_t> stop(numGPUs);
  vector<hipStream_t> stream(numGPUs);

  hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
  cudaCheckError();

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
    hipMalloc(&buffers[d], numElems * sizeof(int));
    cudaCheckError();
    hipMemset(buffers[d], 0, numElems * sizeof(int));
    cudaCheckError();
    hipMalloc(&buffersD2D[d], numElems * sizeof(int));
    cudaCheckError();
    hipMemset(buffersD2D[d], 0, numElems * sizeof(int));
    cudaCheckError();
    hipEventCreate(&start[d]);
    cudaCheckError();
    hipEventCreate(&stop[d]);
    cudaCheckError();
  }

  vector<double> bandwidthMatrix(numGPUs * numGPUs);

  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);

    for (int j = 0; j < numGPUs; j++) {
      int access = 0;
      if (p2p) {
        hipDeviceCanAccessPeer(&access, i, j);
        if (access) {
          hipDeviceEnablePeerAccess(j, 0);
          cudaCheckError();
          hipSetDevice(j);
          cudaCheckError();
          hipDeviceEnablePeerAccess(i, 0);
          cudaCheckError();
          hipSetDevice(i);
          cudaCheckError();
        }
      }

      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      // Block the stream until all the work is queued up
      // DANGER! - hipMemcpy*Async may infinitely block waiting for
      // room to push the operation, so keep the number of repeatitions
      // relatively low.  Higher repeatitions will cause the delay kernel
      // to timeout and lead to unstable results.
      *flag = 0;
      delay<<<1, 1, 0, stream[i]>>>(flag);
      cudaCheckError();
      hipEventRecord(start[i], stream[i]);
      cudaCheckError();

      if (i == j) {
        // Perform intra-GPU, D2D copies
        performP2PCopy(buffers[i], i, buffersD2D[i], i, numElems, nRepeats,
                       access, stream[i]);
      } else {
        if (p2p_method == P2P_WRITE) {
          performP2PCopy(buffers[j], j, buffers[i], i, numElems, nRepeats, access,
                         stream[i]);
        } else {
          performP2PCopy(buffers[i], i, buffers[j], j, numElems, nRepeats, access,
                         stream[i]);
        }
      }

      hipEventRecord(stop[i], stream[i]);
      cudaCheckError();

      // Release the queued events
      *flag = 1;
      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      float time_ms;
      hipEventElapsedTime(&time_ms, start[i], stop[i]);
      double time_s = time_ms / 1e3;

      double gb = numElems * sizeof(int) * nRepeats / (double)1e9;
      if (i == j) {
        gb *= 2;  // must count both the read and the write here
      }
      bandwidthMatrix[i * numGPUs + j] = gb / time_s;
      if (p2p && access) {
        hipDeviceDisablePeerAccess(j);
        hipSetDevice(j);
        hipDeviceDisablePeerAccess(i);
        hipSetDevice(i);
        cudaCheckError();
      }
    }
  }

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipFree(buffers[d]);
    hipFree(buffersD2D[d]);
    cudaCheckError();
    hipEventDestroy(start[d]);
    cudaCheckError();
    hipEventDestroy(stop[d]);
    cudaCheckError();
    hipStreamDestroy(stream[d]);
    cudaCheckError();
  }

  hipHostFree((void *)flag);
  cudaCheckError();

  double* result = new double[bandwidthMatrix.size()];
  copy(bandwidthMatrix.begin(), bandwidthMatrix.end(), result);
  return result;
}

extern "C" double* testBidirectionalBandwidthMatrix(int numElems, int numGPUs, bool p2p,
                                                    unsigned int nWarmups, unsigned int nRepeats) {
  volatile int *flag = NULL;
  vector<int *> buffers(numGPUs);
  vector<int *> buffersD2D(numGPUs);
  vector<hipEvent_t> start(numGPUs);
  vector<hipEvent_t> stop(numGPUs);
  vector<hipStream_t> stream0(numGPUs);
  vector<hipStream_t> stream1(numGPUs);

  hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
  cudaCheckError();

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipMalloc(&buffers[d], numElems * sizeof(int));
    hipMemset(buffers[d], 0, numElems * sizeof(int));
    hipMalloc(&buffersD2D[d], numElems * sizeof(int));
    hipMemset(buffersD2D[d], 0, numElems * sizeof(int));
    cudaCheckError();
    hipEventCreate(&start[d]);
    cudaCheckError();
    hipEventCreate(&stop[d]);
    cudaCheckError();
    hipStreamCreateWithFlags(&stream0[d], hipStreamNonBlocking);
    cudaCheckError();
    hipStreamCreateWithFlags(&stream1[d], hipStreamNonBlocking);
    cudaCheckError();
  }

  vector<double> bandwidthMatrix(numGPUs * numGPUs);

  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);

    for (int j = 0; j < numGPUs; j++) {
      int access = 0;
      if (p2p) {
        hipDeviceCanAccessPeer(&access, i, j);
        if (access) {
          hipSetDevice(i);
          hipDeviceEnablePeerAccess(j, 0);
          cudaCheckError();
          hipSetDevice(j);
          hipDeviceEnablePeerAccess(i, 0);
          cudaCheckError();
        }
      }

      hipSetDevice(i);
      hipStreamSynchronize(stream0[i]);
      hipStreamSynchronize(stream1[j]);
      cudaCheckError();

      // Block the stream until all the work is queued up
      // DANGER! - hipMemcpy*Async may infinitely block waiting for
      // room to push the operation, so keep the number of repeatitions
      // relatively low.  Higher repeatitions will cause the delay kernel
      // to timeout and lead to unstable results.
      *flag = 0;
      hipSetDevice(i);
      // No need to block stream1 since it'll be blocked on stream0's event
      delay<<<1, 1, 0, stream0[i]>>>(flag);
      cudaCheckError();

      // Force stream1 not to start until stream0 does, in order to ensure
      // the events on stream0 fully encompass the time needed for all
      // operations
      hipEventRecord(start[i], stream0[i]);
      hipStreamWaitEvent(stream1[j], start[i], 0);

      if (i == j) {
        // For intra-GPU perform 2 memcopies buffersD2D <-> buffers
        performP2PCopy(buffers[i], i, buffersD2D[i], i, numElems, nRepeats,
                       access, stream0[i]);
        performP2PCopy(buffersD2D[i], i, buffers[i], i, numElems, nRepeats,
                       access, stream1[i]);
      } else {
        if (access && p2p_mechanism == SM) {
          hipSetDevice(j);
        }
        performP2PCopy(buffers[i], i, buffers[j], j, numElems, nRepeats, access,
                       stream1[j]);
        if (access && p2p_mechanism == SM) {
          hipSetDevice(i);
        }
        performP2PCopy(buffers[j], j, buffers[i], i, numElems, nRepeats, access,
                       stream0[i]);
      }

      // Notify stream0 that stream1 is complete and record the time of
      // the total transaction
      hipEventRecord(stop[j], stream1[j]);
      hipStreamWaitEvent(stream0[i], stop[j], 0);
      hipEventRecord(stop[i], stream0[i]);

      // Release the queued operations
      *flag = 1;
      hipStreamSynchronize(stream0[i]);
      hipStreamSynchronize(stream1[j]);
      cudaCheckError();

      float time_ms;
      hipEventElapsedTime(&time_ms, start[i], stop[i]);
      double time_s = time_ms / 1e3;

      double gb = 2.0 * numElems * sizeof(int) * nRepeats / (double)1e9;
      if (i == j) {
        gb *= 2;  // must count both the read and the write here
      }
      bandwidthMatrix[i * numGPUs + j] = gb / time_s;
      if (p2p && access) {
        hipSetDevice(i);
        hipDeviceDisablePeerAccess(j);
        hipSetDevice(j);
        hipDeviceDisablePeerAccess(i);
      }
    }
  }

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipFree(buffers[d]);
    hipFree(buffersD2D[d]);
    cudaCheckError();
    hipEventDestroy(start[d]);
    cudaCheckError();
    hipEventDestroy(stop[d]);
    cudaCheckError();
    hipStreamDestroy(stream0[d]);
    cudaCheckError();
    hipStreamDestroy(stream1[d]);
    cudaCheckError();
  }

  hipHostFree((void *)flag);
  cudaCheckError();

  double* result = new double[bandwidthMatrix.size()];
  copy(bandwidthMatrix.begin(), bandwidthMatrix.end(), result);
  return result;
}

extern "C" double* testLatencyMatrix(int numGPUs, bool p2p, P2PDataTransfer p2p_method,
                                     unsigned int nWarmups, unsigned int nRepeats) {
  int numElems = 4;  // perform 1-int4 transfer.
  volatile int *flag = NULL;
  StopWatchInterface *stopWatch = NULL;
  vector<int *> buffers(numGPUs);
  vector<int *> buffersD2D(numGPUs);  // buffer for D2D, that is, intra-GPU copy
  vector<hipStream_t> stream(numGPUs);
  vector<hipEvent_t> start(numGPUs);
  vector<hipEvent_t> stop(numGPUs);

  hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
  cudaCheckError();

  if (!sdkCreateTimer(&stopWatch)) {
    printf("Failed to create stop watch\n");
    exit(EXIT_FAILURE);
  }
  sdkStartTimer(&stopWatch);

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
    hipMalloc(&buffers[d], sizeof(int) * numElems);
    hipMemset(buffers[d], 0, sizeof(int) * numElems);
    hipMalloc(&buffersD2D[d], sizeof(int) * numElems);
    hipMemset(buffersD2D[d], 0, sizeof(int) * numElems);
    cudaCheckError();
    hipEventCreate(&start[d]);
    cudaCheckError();
    hipEventCreate(&stop[d]);
    cudaCheckError();
  }

  vector<double> gpuLatencyMatrix(numGPUs * numGPUs);
  vector<double> cpuLatencyMatrix(numGPUs * numGPUs);

  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);

    for (int j = 0; j < numGPUs; j++) {
      int access = 0;
      if (p2p) {
        hipDeviceCanAccessPeer(&access, i, j);
        if (access) {
          hipDeviceEnablePeerAccess(j, 0);
          cudaCheckError();
          hipSetDevice(j);
          hipDeviceEnablePeerAccess(i, 0);
          hipSetDevice(i);
          cudaCheckError();
        }
      }
      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      // Block the stream until all the work is queued up
      // DANGER! - hipMemcpy*Async may infinitely block waiting for
      // room to push the operation, so keep the number of repeatitions
      // relatively low.  Higher repeatitions will cause the delay kernel
      // to timeout and lead to unstable results.
      *flag = 0;
      delay<<<1, 1, 0, stream[i]>>>(flag);
      cudaCheckError();
      hipEventRecord(start[i], stream[i]);

      sdkResetTimer(&stopWatch);
      if (i == j) {
        // Perform intra-GPU, D2D copies
        performP2PCopy(buffers[i], i, buffersD2D[i], i, numElems, nRepeats,
                       access, stream[i]);
      } else {
        if (p2p_method == P2P_WRITE) {
          performP2PCopy(buffers[j], j, buffers[i], i, numElems, nRepeats, access,
                         stream[i]);
        } else {
          performP2PCopy(buffers[i], i, buffers[j], j, numElems, nRepeats, access,
                         stream[i]);
        }
      }
      float cpu_time_ms = sdkGetTimerValue(&stopWatch);

      hipEventRecord(stop[i], stream[i]);
      // Now that the work has been queued up, release the stream
      *flag = 1;
      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      float gpu_time_ms;
      hipEventElapsedTime(&gpu_time_ms, start[i], stop[i]);

      gpuLatencyMatrix[i * numGPUs + j] = gpu_time_ms * 1e3 / nRepeats;
      cpuLatencyMatrix[i * numGPUs + j] = cpu_time_ms * 1e3 / nRepeats;
      if (p2p && access) {
        hipDeviceDisablePeerAccess(j);
        hipSetDevice(j);
        hipDeviceDisablePeerAccess(i);
        hipSetDevice(i);
        cudaCheckError();
      }
    }
  }

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipFree(buffers[d]);
    hipFree(buffersD2D[d]);
    cudaCheckError();
    hipEventDestroy(start[d]);
    cudaCheckError();
    hipEventDestroy(stop[d]);
    cudaCheckError();
    hipStreamDestroy(stream[d]);
    cudaCheckError();
  }

  sdkDeleteTimer(&stopWatch);

  hipHostFree((void *)flag);
  cudaCheckError();

  double* result = new double[gpuLatencyMatrix.size()];
  copy(gpuLatencyMatrix.begin(), gpuLatencyMatrix.end(), result);
  return result;
}
