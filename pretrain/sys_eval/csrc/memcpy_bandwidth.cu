/*-----------------------------------------------------------------------*/
/* Program: memcpy_bandwidth                                             */
/* Original code: https://github.com/NVIDIA/cuda-samples                 */
/*                                                                       */
/* This program measures GPU memory transfer rates in GB/s for simple.   */
/*-----------------------------------------------------------------------*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>
#include <memory>


// defines, project
#define CACHE_CLEAR_SIZE (128 * (1e6))  // 128 M

enum memoryMode { PINNED, PAGEABLE };

// CPU cache flush
#define FLUSH_SIZE (256 * 1024 * 1024)
char *flush_buf;

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;


extern "C" float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
    bool wc, unsigned int nWarmups, unsigned int nRepeats);
extern "C" float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
    bool wc, unsigned int nWarmups, unsigned int nRepeats);
extern "C" float testDeviceToDeviceTransfer(unsigned int memSize, 
    unsigned int nWarmups, unsigned int nRepeats);


///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
extern "C" float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                                          bool wc, unsigned int nWarmups, unsigned int nRepeats) {
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  unsigned char *h_idata = NULL;
  unsigned char *h_odata = NULL;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  if (PINNED == memMode) {
  // pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostAlloc((void **)&h_idata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    checkCudaErrors(hipHostMalloc((void **)&h_idata, memSize));
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  } else {
    // pageable memory mode - use malloc
    h_idata = (unsigned char *)malloc(memSize);
    h_odata = (unsigned char *)malloc(memSize);

    if (h_idata == 0 || h_odata == 0) {
      fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
    h_idata[i] = (unsigned char)(i & 0xff);
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // initialize memory
  checkCudaErrors(
    hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));
  
  // warmup
  for (int r = 0; r < nWarmups; r++) {
    checkCudaErrors(
      hipMemcpy(h_odata, d_idata, memSize, hipMemcpyDeviceToHost));
  }

  // copy data from GPU to Host
  if (PINNED == memMode) {
    if (bDontUseGPUTiming) sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < nRepeats; i++) {
      checkCudaErrors(hipMemcpyAsync(h_odata, d_idata, memSize,
                                      hipMemcpyDeviceToHost, 0));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming) {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  } else {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < nRepeats; i++) {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(h_odata, d_idata, memSize, hipMemcpyDeviceToHost));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)nRepeats) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode) {
    checkCudaErrors(hipHostFree(h_idata));
    checkCudaErrors(hipHostFree(h_odata));
  } else {
    free(h_idata);
    free(h_odata);
  }

  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
extern "C" float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                                          bool wc, unsigned int nWarmups, unsigned int nRepeats) {
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;
  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  unsigned char *h_odata = NULL;

  if (PINNED == memMode) {
#if CUDART_VERSION >= 2020
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  } else {
    // pageable memory mode - use malloc
    h_odata = (unsigned char *)malloc(memSize);

    if (h_odata == 0) {
      fprintf(stderr, "Not enough memory available on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  unsigned char *h_cacheClear1 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);
  unsigned char *h_cacheClear2 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);

  if (h_cacheClear1 == 0 || h_cacheClear2 == 0) {
    fprintf(stderr, "Not enough memory available on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
    h_odata[i] = (unsigned char)(i & 0xff);
  }

  for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++) {
    h_cacheClear1[i] = (unsigned char)(i & 0xff);
    h_cacheClear2[i] = (unsigned char)(0xff - (i & 0xff));
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // warmup
  for (int r = 0; r < nWarmups; r++) {
    checkCudaErrors(
      hipMemcpy(d_idata, h_odata, memSize, hipMemcpyHostToDevice));
  }

  // copy host memory to device memory
  if (PINNED == memMode) {
    if (bDontUseGPUTiming) sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < nRepeats; i++) {
      checkCudaErrors(hipMemcpyAsync(d_idata, h_odata, memSize,
                                      hipMemcpyHostToDevice, 0));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming) {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  } else {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < nRepeats; i++) {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(d_idata, h_odata, memSize, hipMemcpyHostToDevice));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)nRepeats) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode) {
    checkCudaErrors(hipHostFree(h_odata));
  } else {
    free(h_odata);
  }

  free(h_cacheClear1);
  free(h_cacheClear2);
  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
extern "C" float testDeviceToDeviceTransfer(unsigned int memSize, unsigned int nWarmups, 
                                            unsigned int nRepeats) {
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  unsigned char *h_idata = (unsigned char *)malloc(memSize);

  if (h_idata == 0) {
    fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the host memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
    h_idata[i] = (unsigned char)(i & 0xff);
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));
  unsigned char *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_odata, memSize));

  // initialize memory
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));

  // warmup
  for (int r = 0; r < nWarmups; r++) {
    checkCudaErrors(
      hipMemcpy(d_odata, d_idata, memSize, hipMemcpyDeviceToDevice));
  }

  // run the memcopy
  sdkStartTimer(&timer);
  checkCudaErrors(hipEventRecord(start, 0));

  for (unsigned int i = 0; i < nRepeats; i++) {
    checkCudaErrors(
        hipMemcpy(d_odata, d_idata, memSize, hipMemcpyDeviceToDevice));
  }

  checkCudaErrors(hipEventRecord(stop, 0));

  // Since device to device memory copies are non-blocking,
  // hipDeviceSynchronize() is required in order to get
  // proper timing.
  checkCudaErrors(hipDeviceSynchronize());

  // get the total elapsed time in ms
  sdkStopTimer(&timer);
  checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

  if (bDontUseGPUTiming) {
    elapsedTimeInMs = sdkGetTimerValue(&timer);
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (2.0f * memSize * (float)nRepeats) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;

  // clean up memory
  sdkDeleteTimer(&timer);
  free(h_idata);
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));

  return bandwidthInGBs;
}